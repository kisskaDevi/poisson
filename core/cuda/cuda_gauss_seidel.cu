#include "hip/hip_runtime.h"
#include "cuda_gauss_seidel.h"

#include "cuda_field.h"

#include <iostream>

#define DEBUG_INFO

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}

namespace cuda {

template<typename type>
__device__ type get_u(cuda::point<type> delta, const cuda::potential<type>& u_field){
    switch (u_field.b_type) {
    case boundary_type::Non:
    case boundary_type::Dirichlet:
        break;
    case boundary_type::Neumann:
        return dot(delta, u_field.derivative);
    }
    return u_field.value;
}

template<typename type>
type* alloc(const type& value){
    type* gpu_p = nullptr;
    checkCudaErrors(hipMalloc((void**)&gpu_p, sizeof(type)));
    checkCudaErrors(hipMemcpy((void*)gpu_p, (void*)&value, sizeof(type), hipMemcpyHostToDevice));
    return gpu_p;
}

template<typename type_gpu, typename type_cpu>
type_gpu* alloc_data(type_cpu** data, size_t n_x, size_t n_y){
    type_gpu* grid_data = nullptr;
    checkCudaErrors(hipMalloc((void**)&grid_data, n_x * n_y * sizeof(type_gpu)));

    type_cpu* cpu_data =  new type_cpu[n_x * n_y];
    for(size_t i = 0; i < n_x; i++){
        for(size_t j = 0; j < n_y; j++){
            cpu_data[i * n_y + j] = data[i][j];
        }
    }
    checkCudaErrors(hipMemcpy((void*)grid_data, (void*)cpu_data, n_x * n_y * sizeof(type_gpu), hipMemcpyHostToDevice));

    return grid_data;
}

template<typename type>
__global__ void calculate_wave(type* dmax, size_t p, size_t q_0, size_t q_n, size_t n_y, const cuda::point<type>* points, const type* func, cuda::potential<type>* u_field){
    int q = q_0 + threadIdx.x + blockIdx.x * blockDim.x;
    if(q > q_n){
        return;
    }

    size_t i = 1 + p - q, j = 1 + q;

    if(u_field[i * n_y + j].b_type == boundary_type::Non){
        type dx = type(0.5) * (points[(i + 1) * n_y + j].x - points[(i - 1) * n_y + j].x);
        type dy = type(0.5) * (points[i * n_y + (j + 1)].y - points[i * n_y + (j - 1)].y);
        type a = type(1) / dx / dx, b = type(1) / dy / dy, c = type(2) * (a + b);

        type temp = u_field[i * n_y + j].value;
        u_field[i * n_y + j].value = (
                                  a * (get_u({ type(2) * dx, 0}, u_field[(i + 1) * n_y + j]) + (u_field[(i + 1) * n_y + j].isNeumann() ? u_field[(i - 1) * n_y + j].value : type(0))) +
                                  a * (get_u({-type(2) * dx, 0}, u_field[(i - 1) * n_y + j]) + (u_field[(i - 1) * n_y + j].isNeumann() ? u_field[(i + 1) * n_y + j].value : type(0))) +
                                  b * (get_u({0,  type(2) * dy}, u_field[i * n_y + (j + 1)]) + (u_field[i * n_y + (j + 1)].isNeumann() ? u_field[i * n_y + (j - 1)].value : type(0))) +
                                  b * (get_u({0, -type(2) * dy}, u_field[i * n_y + (j - 1)]) + (u_field[i * n_y + (j - 1)].isNeumann() ? u_field[i * n_y + (j + 1)].value : type(0))) -
                                  func[i * n_y + j]) / c;

        if(*dmax < std::abs(temp - u_field[i * n_y + j].value)){
            *dmax = std::abs(temp - u_field[i * n_y + j].value);
        }
    }
}

template<typename type>
void gauss_seidel(const gauss_seidel_info<type>& info, const ::points<type>& points, const ::function<type>& func, ::field<type>& u_field){
    cuda::point<type>* grid_data = alloc_data<cuda::point<type>>(points.data, points.n_x, points.n_y);
    type* func_data = alloc_data<type>(func.data, func.n_x, func.n_y);
    cuda::potential<type>* field_data = alloc_data<cuda::potential<type>>(u_field.data, u_field.n_x, u_field.n_y);

    bool convergence = false;
    size_t it = 0;
    for (; !convergence && it < info.max_it; it++ ){
        type* dmax = alloc(type(0));
        for (size_t p = 0; p <= points.n_x - 3 + points.n_y - 3; p++ ){
            calculate_wave<<<10, std::max(points.n_x, points.n_y) / 10 + 1>>>(
                dmax,
                p,
                (p <= points.n_x - 3) ? 0 : p - (points.n_x - 3),
                std::min(p, std::min(points.n_x - 3, points.n_y - 3)),
                points.n_y,
                grid_data,
                func_data,
                field_data);
        }
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipGetLastError());
        type dmax_cpu = info.eps;
        checkCudaErrors(hipMemcpy((void*)&dmax_cpu, (void*)dmax, sizeof(type), hipMemcpyDeviceToHost));
        convergence = dmax_cpu < info.eps;
    }
#ifdef DEBUG_INFO
    std::cout << "   convergence = " << (convergence ? "true" : "false") << std::endl;
    std::cout << "   last_it = " << it << std::endl;
#endif

    ::potential<type>* cpu_data = new ::potential<type>[points.n_x * points.n_y];
    checkCudaErrors(hipMemcpy((void*)cpu_data, (void*)field_data, points.n_x * points.n_y * sizeof(cuda::potential<type>), hipMemcpyDeviceToHost));
    for(size_t i = 0; i < points.n_x; i++){
        for(size_t j = 0; j < points.n_y; j++){
            u_field[i][j] = cpu_data[i * points.n_y + j];
        }
    }
}

template void gauss_seidel<float>(const gauss_seidel_info<float>& info, const ::points<float>& points, const ::function<float>& func, ::field<float>& u_field);
template void gauss_seidel<double>(const gauss_seidel_info<double>& info, const ::points<double>& points, const ::function<double>& func, ::field<double>& u_field);
}
